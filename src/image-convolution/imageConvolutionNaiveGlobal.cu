#include "hip/hip_runtime.h"
#include "image.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

#define FILTER_RADIUS 1
__constant__ float F[2*FILTER_RADIUS+1][2*FILTER_RADIUS+1];

// We assume that the convolutional filter is square
__global__
void naiveImageConvolution(const float *A, float *B, int r, int width , int height)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if ((col < width) && (row < height))
    {
        float Pvalue = 0.f;
        int filter_size = r * 2 + 1;

        for (int i = 0; i < filter_size; ++i)
        {
            for (int j = 0; j < filter_size; ++j)
            {
                int im_row = row - r + i;
                int im_col = col - r + j;
                if ( (im_row >= 0) && (im_row < height) && (im_col >= 0) && (im_col < width))
                {
                    Pvalue += F[i][j] * A[im_row * width + im_col];
                }
            }
        }
        B[row * width + col] = Pvalue;
    }
}

int main() {

    image im = load_image((char*)"../../data/gray.jpg");
    image output = make_image(1, im.h, im.w);

    image sobelXFilter = make_image(1, 3, 3);
    // First row values
    set_pixel(sobelXFilter, 0, 0, 0, -1);
    set_pixel(sobelXFilter, 0, 0, 2, 1);
    // Second row values
    set_pixel(sobelXFilter, 0, 1, 0, -2);
    set_pixel(sobelXFilter, 0, 1, 2, 2);
    // Third row values
    set_pixel(sobelXFilter, 0, 2, 0, -1);
    set_pixel(sobelXFilter, 0, 2, 2, 1);


    size_t inputImgBytes = im.c*im.w*im.h*sizeof(decltype(*im.data));
    size_t inputFilterBytes = sobelXFilter.w*sobelXFilter.h*sizeof(decltype(*sobelXFilter.data));
    size_t outputImgBytes = im.w*im.h*sizeof(decltype(*im.data));

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, inputImgBytes);

    if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, outputImgBytes);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy input filter
    err = hipMemcpyToSymbol(HIP_SYMBOL(F), sobelXFilter.data, inputFilterBytes);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy sobel filter to Global Memory (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_A, im.data, inputImgBytes, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    const dim3 dimGrid((int)ceil((im.w)/4.0), (int)ceil((im.h)/4.0));
	const dim3 dimBlock(4, 4, 1);

    int radious = floor(sobelXFilter.w/2);

    cout << "Radious: " << radious << endl;

    naiveImageConvolution <<< dimGrid, dimBlock  >>> (d_A, d_B, radious, im.w, im.h);

    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    cout << "Copy output data from the CUDA device to the host memory" << endl;

    err = hipMemcpy(output.data, d_B, outputImgBytes, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    feature_normalize(output);
    save_image(output, (char *)"sobelglobal");

    free_image(im);
    free_image(sobelXFilter);
    
    hipFree(d_A);
    hipFree(d_B);


    return 0;
}