#include "hip/hip_runtime.h"
#include "image.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

__global__
void blurImgGray(const float *A, float *B, int width, int height, int kSize, int halfKSzie)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if ((col < width) && (row < height))
    {
        float sumPixel = 0.0f;
        float grayPixel = 0.0f;
        int totalPixels = 0;
        float resultPixel = 0.0f;
        for (int y = -halfKSzie; y != halfKSzie+1; y++) 
        {
            for (int x = -halfKSzie; x != halfKSzie+1; x++)
            {
                if ( (y >= 0) && (y < height) && (x >= 0) && (x < width))
                {
                    grayPixel = A[(row + y)*width + (col + x)];
                    sumPixel += grayPixel;
                    totalPixels++;
                }
            }
        }
        resultPixel = sumPixel / totalPixels;
        B[row * width + col] = resultPixel;
    }
}

int main() {

    image im = load_image((char*)"../../data/gray.jpg");
    image gray = make_image(1, im.h, im.w);

    size_t inputImgBytes = im.c*im.w*im.h*sizeof(decltype(*im.data));
    size_t outputImgBytes = im.w*im.h*sizeof(decltype(*im.data));

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, inputImgBytes);

    if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

      // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, outputImgBytes);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_A, im.data, inputImgBytes, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    const dim3 dimGrid((int)ceil((im.w)/16.0), (int)ceil((im.h)/16.0));
	const dim3 dimBlock(16, 16, 1);

    // Calculate kernel size
    constexpr int KernelSize = 15;
    int kernelCenter = floor(KernelSize/2);

    blurImgGray <<< dimGrid, dimBlock  >>> (d_A, d_B, im.w, im.h, KernelSize, kernelCenter);

    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    cout << "Copy output data from the CUDA device to the host memory" << endl;

    err = hipMemcpy(gray.data, d_B, outputImgBytes, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy vector C from device to host (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    save_image(gray, (char *)"blur");

    return 0;
}